
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <stdlib.h>

#include <math.h>

//declare device and host
//alocate host
//alocate device
//initialise both
//memcopy

__global__ void vecAdd(double *a, double *b, double *c, int n)

{

    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if (id < n)

        c[id] = a[id] + b[id];

}

 

int main( int argc, char* argv[] )

{


    int n = 100;

 


    double *h_a;

    double *h_b;


    double *h_c;


    double *d_a;

    double *d_b;


    double *d_c;

 

    // Allocate memory for each vector on host

    h_a = (double*)malloc(n*sizeof(double));

    h_b = (double*)malloc(n*sizeof(double));

    h_c = (double*)malloc(n*sizeof(double));

 

    // Allocate memory for each vector on GPU

    hipMalloc(&d_a, n*sizeof(double));

    hipMalloc(&d_b, n*sizeof(double));

    hipMalloc(&d_c,n*sizeof(double));

 

    int i;

    // Initialize vectors on host

    for( i = 0; i < n; i++ ) {

        h_a[i] = i;

        h_b[i] = i;

    }

 

    // Copy host vectors to device

    hipMemcpy( d_a, h_a, n*sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy( d_b, h_b, n*sizeof(double), hipMemcpyHostToDevice);


    vecAdd<<<1, 1024>>>(d_a, d_b, d_c, n);

 

    // Copy array back to host

    hipMemcpy( h_c, d_c,n*sizeof(double), hipMemcpyDeviceToHost );


    for(i=0; i<n; i++)

        printf(" %f + %f =%f\n",h_a[i],h_b[i],h_c[i]);


hipDeviceSynchronize();

    hipFree(d_a);

    hipFree(d_b);

    hipFree(d_c);

 
    free(h_a);

    free(h_b);

    free(h_c);

 

    return 0;

}

