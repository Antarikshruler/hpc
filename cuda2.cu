
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void local(float in){
float f;
f=in;
}

__global__ void global(float *a){
a[threadIdx.x]=2.0f*(float)threadIdx.x;
}

__global__ void shared(float *a){
int i,index=threadIdx.x;
float avg,sum=0.0f;
__shared__ float sh_a[128];
sh_a[index]=a[index];
__syncthreads();
//print(avg ther)
}

int main()
{
local<<<1,128>>>(2.0f);

float har[128];
float *dar;

hipMalloc((void **)&dar,sizeof(float)*128);

hipMemcpy((void *)dar,(void *)har,sizeof(float)*128,hipMemcpyHostToDevice);

global<<<1,128>>>(dar);

hipMemcpy((void *)har,(void *)dar,sizeof(float)*128,hipMemcpyDeviceToHost);

shared<<<1,128>>>(dar);

hipMemcpy((void *)har,(void *)dar,sizeof(float)*128,hipMemcpyHostToDevice);

hipDeviceSynchronize();
 return 0;

}
